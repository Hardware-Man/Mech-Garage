/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void strongestNeighborScan_gpu(int * src, int * oldDst, int * newDst, int * oldWeight, int * newWeight, int * madeChanges, int distance, int numEdges) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	for(int i = tid; i < numEdges; i+=total_threads) {
		if(i-distance >= 0) {
			if(src[i] == src[i-distance]) {
				if(oldWeight[i] == oldWeight[i-distance]) {
					if(oldDst[i] < oldDst[i-distance]) {
						newWeight[i] = oldWeight[i];
						newDst[i] = oldDst[i];
					}
					else {
						newWeight[i] = oldWeight[i-distance];
						newDst[i] = oldDst[i-distance];
						madeChanges[0] = 1;
					}
				}
				else if(oldWeight[i] > oldWeight[i-distance]){
					newWeight[i] = oldWeight[i];
					newDst[i] = oldDst[i];
				}
				else {
					newWeight[i] = oldWeight[i-distance];
					newDst[i] = oldDst[i-distance];
					madeChanges[0] = 1;
				}
			}
			else {
				newWeight[i] = oldWeight[i];
				newDst[i] = oldDst[i];
			}
		}
		else {
			newWeight[i] = oldWeight[i];
			newDst[i] = oldDst[i];
		}
	}
}
