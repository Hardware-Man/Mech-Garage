/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void collateSegments_gpu(int * src, int * scanResult, int * output, int numEdges) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	for(int i = tid; i < numEdges; i+=total_threads) {
		if((i+1) < numEdges) {
			if(src[i+1] != src[i]) output[src[i]] = scanResult[i];
		}
		else output[src[i]] = scanResult[i];
	}
}
